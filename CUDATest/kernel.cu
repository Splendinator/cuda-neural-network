#include "hip/hip_runtime.h"
#include "kernel.cuh"






__global__ void randf(float *f, float *min, float *max) {

	int seed1 = (blockIdx.x * blockDim.x + threadIdx.x) & 255;
	int seed2 = ((blockIdx.x * blockDim.x + threadIdx.x) / 256) & 255;

	static const int p[256] =    //Array has loop property. Can keep calling p[p[x]] and loop through all numbers
	{   
		58, 245, 227, 194, 59, 228, 189, 88, 218, 140, 19, 93, 222, 186, 252, 27, 175, 73, 212, 24, 56, 209, 210,
		149, 83, 200, 214, 8, 160, 197, 195, 36, 98, 39, 112, 202, 159, 165, 12, 152, 179, 44, 64, 139, 68, 23,
		239, 91, 102, 211, 226, 141, 60, 190, 72, 111, 45, 134, 171, 30, 231, 48, 104, 246, 78, 114, 106, 207,
		31, 131, 37, 237, 113, 176, 230, 242, 136, 11, 35, 101, 81, 133, 79, 84, 53, 251, 151, 80, 0, 18, 107,
		28, 74, 183, 50, 77, 85, 155, 21, 250, 17, 95, 99, 192, 198, 2, 243, 96, 34, 43, 9, 204, 14, 52, 143,
		70, 6, 42, 205, 236, 148, 232, 201, 213, 49, 108, 10, 138, 92, 119, 193, 4, 105, 125, 181, 219, 216, 51,
		110, 166, 62, 233, 241, 224, 22, 29, 130, 255, 146, 199, 103, 217, 234, 156, 145, 32, 38, 126, 75, 158, 254,
		223, 46, 3, 40, 184, 25, 13, 82, 185, 54, 135, 144, 124, 154, 7, 47, 132, 100, 15, 67, 66, 157, 87, 121, 76,
		180, 220, 20, 203, 229, 127, 225, 109, 33, 122, 215, 235, 150, 97, 94, 89, 168, 120, 196, 206, 164, 153, 247,
		1, 117, 253, 86, 208, 191, 163, 178, 5, 129, 118, 55, 182, 128, 249, 167, 115, 169, 57, 174, 187, 16, 161, 90,
		147, 162, 240, 188, 173, 172, 170, 61, 116, 177, 142, 41, 71, 137, 248, 63, 244, 123, 65, 221, 238, 69, 26 
	};
	
	
	f[blockIdx.x * blockDim.x + threadIdx.x] = (*max - *min) * ((((p[(p[seed1] + seed2)&255] + p[seed2])&255) | (p[(p[seed2] + seed1)&255]) << 8) / 65535.0f) + *min;
	
}


__global__ void add(float *a, float *b) {
	a[blockIdx.x * blockDim.x + threadIdx.x] += b[blockIdx.x * blockDim.x + threadIdx.x];
}

__global__ void total(float *f, int size, float *bias) {

	float *p = f + size*(blockIdx.x * blockDim.x + threadIdx.x);

	while (size > 1) {
		
		if ((size % 2)) {
			p[size - 2] += p[size - 1];
			--size;
			
		}
		size /= 2;
		add <<<1, size>>> (p, p + size);
		
		hipDeviceSynchronize();	//Don't start the next iteration until this one has finished since they depend on each other.

		
	}
	//Sigmoid
	*p = (1 / (1 + powf(E, - (bias[(blockIdx.x * blockDim.x + threadIdx.x)] + *p))));
}

__global__ void mult(float *f, float *m) {
	f[blockIdx.x * blockDim.x + threadIdx.x] *= m[blockIdx.x * blockDim.x + threadIdx.x];
}

__global__ void multNodes(float *weights, float *values, int numNodes) {
	weights[blockIdx.x * blockDim.x + threadIdx.x] *= values[(blockIdx.x * blockDim.x + threadIdx.x) % numNodes] ;
}



